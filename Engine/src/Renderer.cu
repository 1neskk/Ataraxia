#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cfloat>

#include <>

#include "Random.h"
#include "Renderer.h"
#include "BRDF.h"

#include <glm/ext/scalar_constants.hpp>

Renderer::Renderer() : d_spheres_(nullptr), d_materials_(nullptr),d_lights_(nullptr), d_accumulation_(nullptr), h_imageData_(nullptr),
                       d_imageData_(nullptr), m_frameIndex(1)
{}

Renderer::~Renderer()
{
	hipFree(d_imageData_);
	hipFree(d_accumulation_);
	freeDeviceMemory();

    delete[] h_imageData_;
}

void Renderer::onResize(uint32_t width, uint32_t height)
{
	if (m_image && m_image->getWidth() == width && m_image->getHeight() == height)
		return;

	m_image = std::make_shared<Image>(width, height, ImageType::RGBA);

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[width * height];

    hipFree(d_imageData_);
    hipMalloc(&d_imageData_, static_cast<unsigned long long>(width) * height * sizeof(uint32_t));

	hipFree(d_accumulation_);
	hipMalloc(&d_accumulation_, static_cast<unsigned long long>(width) * height * sizeof(glm::vec4));

    m_width = width;
    m_height = height;

    m_frameIndex = 1;
}

void Renderer::Render(Camera& camera, const Scene& scene)
{
    m_scene = &scene;

	allocateDeviceMemory(scene);

    if (m_frameIndex == 1)
		hipMemset(d_accumulation_, 0, static_cast<unsigned long long>(m_width) * m_height * sizeof(glm::vec4));

    if (!m_image)
        return;

	DeviceCamera d_camera;
    camera.allocateDevice(d_camera);

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

	kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, d_spheres_, scene.spheres.size(), d_camera,
        d_materials_, scene.materials.size(), d_accumulation_, m_frameIndex, d_lights_, scene.lights.size());

	hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
		std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << "\n";
        return;
    }

    hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
        std::cerr << "CUDA kernel synchronization error: " << hipGetErrorString(err) << "\n";
		return;
	}

    err = hipMemcpy(h_imageData_, d_imageData_, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
	    std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
		return;
	}

    m_image->setData(h_imageData_);
	camera.freeDevice(d_camera);

    if (m_settings.accumulation)
        m_frameIndex++;
    else
        m_frameIndex = 1;
}

void Renderer::allocateDeviceMemory(const Scene& scene)
{
	hipFree(d_spheres_);
	hipMalloc(&d_spheres_, scene.spheres.size() * sizeof(Sphere));
	hipMemcpy(d_spheres_, scene.spheres.data(), scene.spheres.size() * sizeof(Sphere), hipMemcpyHostToDevice);
	

	hipFree(d_materials_);
	hipMalloc(&d_materials_, scene.materials.size() *sizeof(Material));
    hipMemcpy(d_materials_, scene.materials.data(), scene.materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipFree(d_lights_);
	hipMalloc(&d_lights_, scene.lights.size() * sizeof(Light));
	hipMemcpy(d_lights_, scene.lights.data(), scene.lights.size() * sizeof(Light), hipMemcpyHostToDevice);
}

void Renderer::freeDeviceMemory()
{
	hipFree(d_spheres_);
	d_spheres_ = nullptr;

    hipFree(d_materials_);
	d_materials_ = nullptr;

	hipFree(d_lights_);
	d_lights_ = nullptr;

	hipFree(d_accumulation_);
	d_accumulation_ = nullptr;
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera d_camera, const Material* materials, size_t numMaterials, glm::vec4* accumulation,
	uint32_t frameIndex, const Light* lights, size_t numLights)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
	{
		const glm::vec4 color = Renderer::perPixel(x, y, width, spheres, numSpheres, d_camera, materials, numMaterials, frameIndex, lights,
			numLights);
		const uint32_t pixelIndex = x + y * width;
        accumulation[pixelIndex] += color;
		glm::vec4 finalColor = accumulation[pixelIndex] / static_cast<float>(frameIndex);
        finalColor = glm::clamp(finalColor, 0.0f, 1.0f);
		imageData[pixelIndex] = colorUtils::vec4ToRGBA(finalColor);
    }
}

__device__ Renderer::HitRecord Renderer::traceRay(const Ray& ray, const Sphere* spheres, size_t numSpheres)
{
    int closestSphere = -1;
    float tmin = FLT_MAX;

	for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& [center, radius, id] = spheres[i];

		glm::vec3 oc = ray.origin - center;

		const float a = glm::dot(ray.direction, ray.direction);
		const float b = 2.0f * glm::dot(oc, ray.direction);
		const float c = glm::dot(oc, oc) - radius * radius;
		const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
			continue;

		float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
		float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
		const float t = t0 < t1 ? t0 : t1;

		if (t > 0.0f && t < tmin)
        {
			tmin = t;
            closestSphere = static_cast<int>(i);
		}
	}

    if (closestSphere < 0)
        return rayMiss(ray);

	return rayHit(ray, tmin, closestSphere, spheres);
}

__device__ glm::vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera& d_camera, const Material* materials, size_t numMaterials, uint32_t frameIndex,
	const Light* lights, size_t numLights)
{
	__shared__ Light sharedLights[10];

	if (threadIdx.x < numLights)
		sharedLights[threadIdx.x] = lights[threadIdx.x];

	__syncthreads();

	Ray ray;
	ray.origin = d_camera.position;
	ray.direction = d_camera.rayDirection[x + y * width];

    glm::vec3 color(0.0f);
    glm::vec3 throughput(1.0f);

	uint32_t seed = x + y * width;
	seed *= frameIndex;

	constexpr int bounces = 5; // Max bounces

	for (int i = 0; i < bounces; i++)
	{
		seed += i;

		HitRecord ht = traceRay(ray, spheres, numSpheres);
		if (ht.t < 0.0f)
		{
#define SKY_LIGHT 0
#if SKY_LIGHT
			glm::vec3 missColor(0.6f, 0.7f, 0.9f);
			color += missColor * throughput;
			break;
#else
			color += glm::vec3(0.0f);
			break;
#endif
		}

		const auto& [center, radius, id] = spheres[ht.id];
		const Material* mat = &materials[id];

		glm::vec3 baseReflectivity = glm::mix(mat->F0, mat->albedo, mat->metallic);
		for (size_t j = 0; j < numLights && j < 10; j++)
		{
			const Light& light = sharedLights[j];
			glm::vec3 L = light.position - ht.worldPos;
			const float distance = glm::length(L);
			L = glm::normalize(L);

			Ray shadowRay;
			shadowRay.origin = ht.worldPos + ht.worldNormal * 0.0001f;
			shadowRay.direction = L;

			HitRecord shadowHt = traceRay(shadowRay, spheres, numSpheres);
			if (shadowHt.t > 0.0f && shadowHt.t < distance)
				continue;

			const glm::vec3 V = -ray.direction;
			const glm::vec3 N = ht.worldNormal;

			const glm::vec3 diffuse = BRDF::lambertian(mat->albedo);
			const glm::vec3 specular = BRDF::cookTorrance(mat->albedo, baseReflectivity, mat->metallic, mat->roughness, N, V, L);

			const glm::vec3 brdf = diffuse + specular;
			const float NdotL = glm::max(glm::dot(N, L), 0.0f);

			color += light.intensity * brdf * NdotL * throughput;
		}

		throughput *= mat->albedo;

		ray.origin = ht.worldPos + ht.worldNormal * 0.0001f;

		if (mat->metallic > 0.0f)
			ray.direction = BRDF::sampleGGX(ht.worldNormal, mat->roughness, seed);
		else
			ray.direction = BRDF::sampleHemisphereCosineWeighted(ht.worldNormal, seed);
	}
    return { color, 1.0f };
}

__device__ Renderer::HitRecord Renderer::rayMiss(const Ray& ray)
{
    HitRecord ht;
	ht.t = -1.0f;
    return ht;
}

__device__ Renderer::HitRecord Renderer::rayHit(const Ray& ray, float tmin, const int index, const Sphere* spheres)
{
    HitRecord ht;
    ht.t = tmin;
    ht.id = index;

	const glm::vec3 origin = ray.origin - spheres[index].center;
    ht.worldPos = origin + ray.direction * tmin;
	ht.worldNormal = glm::normalize(ht.worldPos);

	ht.worldPos += spheres[index].center;

    return ht;
}