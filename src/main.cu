#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define _USE_MATH_DEFINES
#include <cmath>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../thirdparty/stb/stb_image_write.h"

struct Vec3
{
	float x, y, z;
	__host__ __device__ Vec3() : x(0.0f), y(0.0f), z(0.0f) {}
	__host__ __device__ Vec3(float x, float y, float z) : x(x), y(y), z(z) {}

	__host__ __device__ Vec3 operator+(const Vec3& v) const { return Vec3(x + v.x, y + v.y, z + v.z); }
	__host__ __device__ Vec3 operator-(const Vec3& v) const { return Vec3(x - v.x, y - v.y, z - v.z); }
	__host__ __device__ Vec3 operator*(float s) const { return Vec3(x * s, y * s, z * s); }
	__host__ __device__ Vec3 operator/(float s) const { return Vec3(x / s, y / s, z / s); }

    // scalar
    __host__ __device__ Vec3 operator*(const Vec3& v) const { return Vec3(x * v.x, y * v.y, z * v.z); }
    __host__ __device__ Vec3 operator/(const Vec3& v) const { return Vec3(x / v.x, y / v.y, z / v.z); }

	__host__ __device__ float length() const { return sqrtf(x * x + y * y + z * z); }
	__host__ __device__ Vec3 normalize() const { return *this / length(); }
	__host__ __device__ static float dot(const Vec3& a, const Vec3& b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
	
	__host__ __device__ static Vec3 cross(const Vec3& a, const Vec3& b)
	{ return Vec3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x); }

};

struct Ray
{
    Vec3 origin;
    Vec3 direction;
};

struct Sphere
{
    Vec3 center;
    float radius;
};

__device__ bool intersect(const Ray& ray, const Sphere& sphere, float& t)
{
    const Vec3 oc = ray.origin - sphere.center;
    const float a = Vec3::dot(ray.direction, ray.direction);
    const float b = 2.0f * Vec3::dot(oc, ray.direction);
    const float c = Vec3::dot(oc, oc) - sphere.radius * sphere.radius;
    const float discriminant = b * b - 4.0f * a * c;

    if (discriminant < 0.0f)
    {
        t = (-b + sqrtf(discriminant)) / (2.0f * a);
        return t >= 0.0f;
    }
    else
    {
        t = (-b - sqrtf(discriminant)) / (2.0f * a);
        return t > 0.0f;
    }
}

// CUDA kernel
__global__ void renderKernel(Vec3* image, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    float u = static_cast<float>(x) / static_cast<float>(width);
    float v = static_cast<float>(y) / static_cast<float>(height);

    Ray ray = { Vec3(0.0f, 0.0f, 0.0f), Vec3(2.0f * u - 1.0f, 2.0f * v - 1.0f, -1.0f).normalize() };
    Sphere sphere = { Vec3(0.0f, 0.0f, -3.0f), 1.0f };

    float t = INFINITY;
    if (intersect(ray, sphere, t))
    {
        image[idx] = Vec3(1.0f, 0.0f, 0.0f);
    }
    else
    {
        image[idx] = Vec3(0.0f, 0.0f, 0.0f);
    }
}

int main()
{
    constexpr int width = 1200;
    constexpr int height = 800;
    constexpr size_t imageSize = width * height * sizeof(Vec3);

    auto* h_image = static_cast<Vec3*>(malloc(imageSize));
    Vec3* d_image;
    hipMalloc((void**)&d_image, imageSize);

    Sphere h_spheres[1];
    h_spheres[0].center = Vec3(0.0f, 0.0f, 0.0f);
    h_spheres[0].radius = 1.0f;

    Sphere* d_spheres;
    hipMalloc((void**)&d_spheres, sizeof(h_spheres));
    hipMemcpy(d_spheres, h_spheres, sizeof(h_spheres), hipMemcpyHostToDevice);

    // Debugging: Print out the sphere data
    std::cout << "Sphere center: " << h_spheres[0].center.x << ", " << h_spheres[0].center.y << ", " << h_spheres[0].center.z << std::endl;
    std::cout << "Sphere radius: " << h_spheres[0].radius << std::endl;

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    renderKernel<<<gridSize, blockSize>>>(d_image, width, height);

    hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);

    // Debugging: Check if the image data is correctly generated
    int hitCount = 0;
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            const int idx = y * width + x;
            const Vec3 color = h_image[idx];
            if (color.x > 0.0f)
                hitCount++;
        }
    }
    std::cout << "Number of hits: " << hitCount << std::endl;

    const auto image_data = static_cast<unsigned char*>(malloc(width * height * 4));
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            const int idx = y * width + x;
            const Vec3 color = h_image[idx];
            image_data[4 * idx + 0] = static_cast<unsigned char>(color.x * 255.0f);
            image_data[4 * idx + 1] = static_cast<unsigned char>(color.y * 255.0f);
            image_data[4 * idx + 2] = static_cast<unsigned char>(color.z * 255.0f);
            image_data[4 * idx + 3] = 255; // Alpha
        }
    }

    stbi_write_png("output.png", width, height, 4, image_data, width * 4);
    std::cout << "Image saved to 'output.png'" << std::endl;

    hipFree(d_image);
    hipFree(d_spheres);

    free(h_image);
    free(image_data);

    return 0;
}
