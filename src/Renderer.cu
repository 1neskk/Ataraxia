#include "hip/hip_runtime.h"
#include "Renderer.h"

#include <algorithm>
#include <>

constexpr auto pi = 3.14159265358979323846f;

Renderer::Renderer() : m_image(nullptr)
{}

Renderer::~Renderer()
{
    if (d_imageData_)
        hipFree(d_imageData_);

    delete[] h_imageData_;
}

void Renderer::onResize(uint32_t width, uint32_t height)
{
    if (m_image)
    {
        if (m_image->getWidth() == width && m_image->getHeight() == height)
        {
            return;
        }
        m_image->resize(width, height);
    }
    else
        m_image = std::make_shared<Image>(width, height, ImageType::RGBA);

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[static_cast<uint64_t>(width) * height];

    if (d_imageData_)
        hipFree(d_imageData_);

    hipMalloc(&d_imageData_, static_cast<uint64_t>(width) * height * sizeof(uint32_t));

    m_width = width;
    m_height = height;
}

void Renderer::Render(const Scene& scene)
{
    m_scene = &scene;

    if (!m_image)
        return;

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

    kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, m_scene->spheres, m_scene->numSpheres);

    hipDeviceSynchronize();

    hipMemcpy(h_imageData_, d_imageData_, static_cast<uint64_t>(m_width) * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    m_image->setData(h_imageData_);
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres, size_t numSpheres)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        const Vec4 color = Renderer::perPixel(x, y, width, height, spheres, numSpheres);
        imageData[x + y * width] = colorUtils::vec4ToRGBA(color);
    }
}

__device__ bool Renderer::intersect(const Ray& ray, const Sphere* spheres, size_t numSpheres, float& t)
{
    int closestSphere = -1;
    float tmin = INFINITY;

    for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& [center, radius, id] = spheres[i];

        Vec3 origin = ray.origin - center;

        const float a = ray.direction.dot(ray.direction);
        const float b = 2.0f * origin.dot(ray.direction);
        const float c = origin.dot(origin) - radius * radius;
        const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
            continue;

        float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
        float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
		t = t0 < t1 ? t0 : t1;

        if (t > 0.0f && t < tmin)
        {
            tmin = t;
            closestSphere = static_cast<int>(i);
        }
    }

    if (closestSphere < 0)
        return false;
    
    t = tmin;
    return true;
}

__device__ Vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, uint32_t height,
                                   const Sphere* spheres, size_t numSpheres)
{
    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);
    constexpr float fov = 90.0f;
    const float scale = tan(fov * 0.5f * pi / 180.0f);

    const float u = (2.0f * (static_cast<float>(x) + 0.5f) / static_cast<float>(width) - 1.0f) * aspectRatio * scale;
    const float v = (1.0f - 2.0f * (static_cast<float>(y) + 0.5f) / static_cast<float>(height)) * scale;

    const Vec3 direction = Vec3(u, v, -1.0f).normalize();
    const Ray ray = { Vec3(0.0f, 0.0f, 0.0f), direction };

    float t;
    if (intersect(ray, spheres, numSpheres, t))
    {
    	const auto& [center, radius, id] = spheres[0];
		const Vec3 hitPoint = ray.origin + ray.direction * t;
		const Vec3 normal = (hitPoint - center).normalize();

		return { normal.x, normal.y, normal.z, 1.0f };
    }

    return { 0.0f, 0.0f, 0.0f, 1.0f };
}
