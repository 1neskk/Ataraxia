#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cfloat>

#include "Renderer.h"

Renderer::Renderer() : d_spheres_(nullptr), h_imageData_(nullptr), d_imageData_(nullptr)
{}

Renderer::~Renderer()
{
    if (d_imageData_)
        hipFree(d_imageData_);

    if (d_spheres_)
        freeDeviceMemory();

    delete[] h_imageData_;
}

void Renderer::onResize(uint32_t width, uint32_t height)
{
    if (m_image)
    {
        if (m_image->getWidth() == width && m_image->getHeight() == height)
        {
            return;
        }
        m_image->resize(width, height);
    }
    else
    {
        m_image = std::make_shared<Image>(width, height, ImageType::RGBA);
    }

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[width * height];

    if (d_imageData_)
        hipFree(d_imageData_);

    hipError_t err = hipMalloc(&d_imageData_, width * height * sizeof(uint32_t));
    if (err != hipSuccess)
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";

    m_width = width;
    m_height = height;
}

void Renderer::Render(Camera& camera, const Scene& scene)
{
    m_scene = &scene;

	allocateDeviceMemory(scene);

    if (!m_image)
        return;

	DeviceCamera d_camera;
    camera.allocateDevice(d_camera);

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

	kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, d_spheres_, scene.spheres.size(), d_camera);

	hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
		std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << "\n";
        return;
    }

    hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
        std::cerr << "CUDA kernel synchronization error: " << hipGetErrorString(err) << "\n";
		return;
	}

    err = hipMemcpy(h_imageData_, d_imageData_, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
	    std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
		return;
	}

    m_image->setData(h_imageData_);
	camera.freeDevice(d_camera);
}

void Renderer::allocateDeviceMemory(const Scene& scene)
{
	if (d_spheres_)
		hipFree(d_spheres_);

	size_t numSpheres = scene.spheres.size();
	hipError_t err = hipMalloc(&d_spheres_, numSpheres * sizeof(Sphere));
	if (err != hipSuccess)
		std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";

	err = hipMemcpy(d_spheres_, scene.spheres.data(), numSpheres * sizeof(Sphere), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
}

void Renderer::freeDeviceMemory()
{
	if (d_spheres_)
	{
        hipFree(d_spheres_);
        d_spheres_ = nullptr;
	}
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera d_camera)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
	{
		const glm::vec4 color = Renderer::perPixel(x, y, width, spheres, numSpheres, d_camera);
		imageData[x + y * width] = colorUtils::vec4ToRGBA(color);
    }
}

__device__ Renderer::HitRecord Renderer::traceRay(const Ray& ray, const Sphere* spheres, size_t numSpheres)
{
    int closestSphere = -1;
    float tmin = FLT_MAX;

	for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& [center, radius, id] = spheres[i];

		glm::vec3 oc = ray.origin - center;

		const float a = glm::dot(ray.direction, ray.direction);
		const float b = 2.0f * glm::dot(oc, ray.direction);
		const float c = glm::dot(oc, oc) - radius * radius;
		const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
			continue;

		float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
		float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
		const float t = t0 < t1 ? t0 : t1;

		if (t > 0.0f && t < tmin)
        {
	        tmin = t;
            closestSphere = static_cast<int>(i);
		}
	}

    if (closestSphere < 0)
        return rayMiss(ray);

	return rayHit(ray, tmin, closestSphere, spheres);
}

__device__ glm::vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera& d_camera)
{
    Ray ray;
    ray.origin = d_camera.position;
    ray.direction = d_camera.rayDirection[x + y * width];

    glm::vec3 color(0.0f);
    float m = 1.0f;
    constexpr int bounces = 1;

    for (int i = 0; i < bounces; i++)
    {
		HitRecord ht = traceRay(ray, spheres, numSpheres);
        if (ht.t < 0.0f)
        {
			glm::vec3 missColor(0.6f, 0.7f, 0.9f);
			color += missColor * m;
            break;
        }

        glm::vec3 lightDir = glm::normalize(glm::vec3(-1.0f, -1.0f, -1.0f));
		float light = glm::dot(-lightDir, ht.normal);
		if (light < 0.0f)
			light = 0.0f;

        const auto& [center, radius, id] = spheres[ht.id];
		//const Material& mat = m_scene->materials[id];

        glm::vec3 sphereColor = glm::vec3(1.0f, 0.0f, 0.0f) * light;
		color += sphereColor * m;

        m *= 0.5f;

		ray.origin = ht.worldNormal + ht.normal * 0.0001f;
		ray.direction = glm::reflect(ray.direction, ht.normal * glm::vec3(0.5f, 0.5f, 0.5f));
    }
    return { color, 1.0f };
}

__device__ Renderer::HitRecord Renderer::rayMiss(const Ray& ray)
{
    HitRecord ht;
	ht.t = -1.0f;
    return ht;
}

__device__ Renderer::HitRecord Renderer::rayHit(const Ray& ray, float tmin, int index, const Sphere* spheres)
{
    HitRecord ht;
    ht.t = tmin;
    ht.id = index;

	const glm::vec3 origin = ray.origin - spheres[index].center;
    ht.worldNormal = origin + ray.direction * tmin;
	ht.normal = glm::normalize(ht.worldNormal);

	ht.worldNormal += spheres[index].center;

    return ht;
}