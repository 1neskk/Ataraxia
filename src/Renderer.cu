#include "hip/hip_runtime.h"
#include "Renderer.h"

#include <algorithm>

Renderer::Renderer() : m_image(nullptr), h_imageData_(nullptr), d_imageData_(nullptr)
{}

Renderer::~Renderer()
{
    if (d_imageData_)
        hipFree(d_imageData_);

    delete[] h_imageData_;
}


void Renderer::onResize(uint32_t width, uint32_t height)
{
    if (m_image)
    {
        if (m_image->getWidth() == width && m_image->getHeight() == height)
        {
            return;
        }
        m_image->resize(width, height);
    }
    else
        m_image = std::make_shared<Image>(width, height, ImageType::RGBA);

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[width * height];

    if (d_imageData_)
        hipFree(d_imageData_);

    hipMalloc(&d_imageData_, width * height * sizeof(uint32_t));

    m_width = width;
    m_height = height;
}

void Renderer::Render(const Scene& scene)
{
    m_scene = &scene;

    if (!m_image)
        return;

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

    kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, m_scene->spheres, m_scene->numSpheres);

    hipMemcpy(h_imageData_, d_imageData_, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    m_image->setData(h_imageData_);
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres, size_t numSpheres)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        Vec4 color = Renderer::perPixel(x, y, width, height, spheres, numSpheres);
        imageData[x + y * width] = colorUtils::vec4ToRGBA(color);
    }
}

__device__ bool Renderer::intersect(const Ray& ray, const Sphere* spheres, size_t numSpheres, float& t)
{
    int closestSphere = -1;
    float tmin = INFINITY;

    for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& sphere = spheres[i];

        Vec3 origin = ray.origin - sphere.center;

        const float a = ray.direction.dot(ray.direction);
        const float b = 2.0f * origin.dot(ray.direction);
        const float c = origin.dot(origin) - sphere.radius * sphere.radius;
        const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
            continue;

        float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
        float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
        const float t = t0 < t1 ? t0 : t1;

        if (t > 0.0f && t < tmin)
        {
            tmin = t;
            closestSphere = static_cast<int>(i);
        }
    }

    if (closestSphere < 0)
        return false;
    
    t = tmin;
    return true;
}

__device__ Vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, uint32_t height,
                                   const Sphere* spheres, size_t numSpheres)
{
    const float aspectRatio = width / static_cast<float>(height);
    const float fov = 90.0f;
    const float scale = tan(fov * 0.5f * M_PI / 180.0f);

    const float u = (2.0f * (x + 0.5f) / width - 1.0f) * aspectRatio * scale;
    const float v = (1.0f - 2.0f * (y + 0.5f) / height) * scale;

    Vec3 direction = Vec3(u, v, -1.0f).normalize();
    Ray ray = { Vec3(0.0f, 0.0f, 0.0f), direction };

    float t;
    if (intersect(ray, spheres, numSpheres, t))
    {
        return Vec4(1.0f, 0.0f, 0.0f, 1.0f);
    }

    return Vec4(0.0f, 0.0f, 0.0f, 1.0f);
}
