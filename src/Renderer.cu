#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cfloat>

#include "Random.h"
#include "Renderer.h"

Renderer::Renderer() : d_spheres_(nullptr), d_materials_(nullptr), d_accumulation_(nullptr), h_imageData_(nullptr),
						d_imageData_(nullptr), m_frameIndex(1)
{}

Renderer::~Renderer()
{
    if (d_imageData_)
        hipFree(d_imageData_);

    if (d_accumulation_)
		hipFree(d_accumulation_);

    if (d_spheres_)
        freeDeviceMemory();

    delete[] h_imageData_;
}

void Renderer::onResize(uint32_t width, uint32_t height)
{
    if (m_image)
    {
        if (m_image->getWidth() == width && m_image->getHeight() == height)
        {
            return;
        }
        m_image->resize(width, height);
    }
    else
        m_image = std::make_shared<Image>(width, height, ImageType::RGBA);

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[width * height];

    if (d_imageData_)
        hipFree(d_imageData_);

    hipError_t err = hipMalloc(&d_imageData_, static_cast<unsigned long long>(width) * height * sizeof(uint32_t));
    if (err != hipSuccess)
        std::cerr << "hipMalloc failed for d_imageData_:" << hipGetErrorString(err) << "\n";

    if (d_accumulation_)
		hipFree(d_accumulation_);

	err = hipMalloc(&d_accumulation_, static_cast<unsigned long long>(width) * height * sizeof(glm::vec4));
    if (err != hipSuccess)
		std::cerr << "hipMalloc failed for d_accumulation_:" << hipGetErrorString(err) << "\n";

    m_width = width;
    m_height = height;

    m_frameIndex = 1;
}

void Renderer::Render(Camera& camera, const Scene& scene)
{
    m_scene = &scene;

	allocateDeviceMemory(scene);

    if (m_frameIndex == 1)
		hipMemset(d_accumulation_, 0, static_cast<unsigned long long>(m_width) * m_height * sizeof(glm::vec4));

    if (!m_image)
        return;

	DeviceCamera d_camera;
    camera.allocateDevice(d_camera);

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

	kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, d_spheres_, scene.spheres.size(), d_camera,
        d_materials_, d_accumulation_, m_frameIndex);

	hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
		std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << "\n";
        return;
    }

    hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
        std::cerr << "CUDA kernel synchronization error: " << hipGetErrorString(err) << "\n";
		return;
	}

    err = hipMemcpy(h_imageData_, d_imageData_, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
	    std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
		return;
	}

    m_image->setData(h_imageData_);
	camera.freeDevice(d_camera);

    if (m_settings.accumulation)
        m_frameIndex++;
    else
        m_frameIndex = 1;
}

void Renderer::allocateDeviceMemory(const Scene& scene)
{
	if (d_spheres_)
		hipFree(d_spheres_);

	size_t numSpheres = scene.spheres.size();
	hipError_t err = hipMalloc(&d_spheres_, numSpheres * sizeof(Sphere));
	if (err != hipSuccess)
		std::cerr << "hipMalloc failed for d_spheres_: " << hipGetErrorString(err) << "\n";

	err = hipMemcpy(d_spheres_, scene.spheres.data(), numSpheres * sizeof(Sphere), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		std::cerr << "hipMemcpy failed for d_spheres_: " << hipGetErrorString(err) << "\n";

	if (d_materials_)
		hipFree(d_materials_);

	size_t numMaterials = scene.materials.size();
    err = hipMalloc(&d_materials_, numMaterials * sizeof(Material));
    if (err != hipSuccess)
		std::cerr << "hipMalloc failed for d_materials_: " << hipGetErrorString(err) << "\n";

    err = hipMemcpy(d_materials_, scene.materials.data(), numMaterials * sizeof(Material), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		std::cerr << "hipMemcpy failed for d_materials_: " << hipGetErrorString(err) << "\n";
}

void Renderer::freeDeviceMemory()
{
	if (d_spheres_)
	{
		hipFree(d_spheres_);
		d_spheres_ = nullptr;
	}

	if (d_materials_)
    {
	    hipFree(d_materials_);
		d_materials_ = nullptr;
	}

	if (d_accumulation_)
	{
		hipFree(d_accumulation_);
		d_accumulation_ = nullptr;
	}
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera d_camera, const Material* materials, glm::vec4* accumulation, uint32_t frameIndex)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
	{
		const glm::vec4 color = Renderer::perPixel(x, y, width, spheres, numSpheres, d_camera, materials, frameIndex);

		const uint32_t pixelIndex = x + y * width;
        glm::vec4 finalColor = accumulation[pixelIndex];

		finalColor = (finalColor * static_cast<float>(frameIndex) + color) / static_cast<float>(frameIndex + 1);
		accumulation[pixelIndex] = finalColor;

		imageData[pixelIndex] = colorUtils::vec4ToRGBA(finalColor);
    }
}

__device__ Renderer::HitRecord Renderer::traceRay(const Ray& ray, const Sphere* spheres, size_t numSpheres)
{
    int closestSphere = -1;
    float tmin = FLT_MAX;

	for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& [center, radius, id] = spheres[i];

		glm::vec3 oc = ray.origin - center;

		const float a = glm::dot(ray.direction, ray.direction);
		const float b = 2.0f * glm::dot(oc, ray.direction);
		const float c = glm::dot(oc, oc) - radius * radius;
		const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
			continue;

		float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
		float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
		const float t = t0 < t1 ? t0 : t1;

		if (t > 0.0f && t < tmin)
        {
	        tmin = t;
            closestSphere = static_cast<int>(i);
		}
	}

    if (closestSphere < 0)
        return rayMiss(ray);

	return rayHit(ray, tmin, closestSphere, spheres);
}

__device__ glm::vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, const Sphere* spheres,
    size_t numSpheres, const DeviceCamera& d_camera, const Material* materials, uint32_t frameIndex)
{
    Ray ray;
    ray.origin = d_camera.position;
    ray.direction = d_camera.rayDirection[x + y * width];

    glm::vec3 color(0.0f);
    float m = 1.0f;
    constexpr int bounces = 10;

    uint32_t seed = x + y * width;
	seed *= frameIndex;

    for (int i = 0; i < bounces; i++)
    {
        seed += i;

		HitRecord ht = traceRay(ray, spheres, numSpheres);
        if (ht.t < 0.0f)
        {
			glm::vec3 missColor(0.6f, 0.7f, 0.9f);
			color += missColor * m;
            break;
        }

        glm::vec3 lightDir = glm::normalize(glm::vec3(-1.0f, -1.0f, -1.0f));
		float light = glm::dot(-lightDir, ht.normal);
		if (light < 0.0f)
			light = 0.0f;

        const auto& [center, radius, id] = spheres[ht.id];
		const Material& mat = materials[id];

        glm::vec3 sphereColor = mat.albedo * light;
		color += sphereColor * m;

        m *= 0.5f;

		ray.origin = ht.worldNormal + ht.normal * 0.0001f;
        ray.direction = glm::reflect(ray.direction, ht.normal * Random::Random::PcgInUnitSphere(seed));
    }
    return { color, 1.0f };
}

__device__ Renderer::HitRecord Renderer::rayMiss(const Ray& ray)
{
    HitRecord ht;
	ht.t = -1.0f;
    return ht;
}

__device__ Renderer::HitRecord Renderer::rayHit(const Ray& ray, float tmin, int index, const Sphere* spheres)
{
    HitRecord ht;
    ht.t = tmin;
    ht.id = index;

	const glm::vec3 origin = ray.origin - spheres[index].center;
    ht.worldNormal = origin + ray.direction * tmin;
	ht.normal = glm::normalize(ht.worldNormal);

	ht.worldNormal += spheres[index].center;

    return ht;
}