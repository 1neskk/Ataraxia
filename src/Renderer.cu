#include "hip/hip_runtime.h"
#include "Renderer.h"

#include <algorithm>
#include <iostream>
#include <cfloat>

Renderer::Renderer() : d_spheres_(nullptr), h_imageData_(nullptr), d_imageData_(nullptr)
{}

Renderer::~Renderer()
{
    if (d_imageData_)
        hipFree(d_imageData_);

    if (d_spheres_)
        freeDeviceMemory();

    delete[] h_imageData_;
}

void Renderer::onResize(uint32_t width, uint32_t height)
{
    if (m_image)
    {
        if (m_image->getWidth() == width && m_image->getHeight() == height)
        {
            return;
        }
        m_image->resize(width, height);
    }
    else
    {
        m_image = std::make_shared<Image>(width, height, ImageType::RGBA);
    }

    delete[] h_imageData_;
    h_imageData_ = new uint32_t[width * height];

    if (d_imageData_)
        hipFree(d_imageData_);

    hipError_t err = hipMalloc(&d_imageData_, width * height * sizeof(uint32_t));
    if (err != hipSuccess)
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";

    m_width = width;
    m_height = height;
}

void Renderer::Render(const Camera& camera, const Scene& scene)
{
    m_scene = &scene;

	allocateDeviceMemory(scene);

    if (!m_image)
        return;

    glm::vec3* d_rayDirs;
	size_t rayDirSize = m_width * m_height * sizeof(glm::vec3);
    hipMalloc(&d_rayDirs, rayDirSize);
    hipMemcpy(d_rayDirs, camera.getRayDirection().data(), rayDirSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 numBlocks((m_width + blockSize.x - 1) / blockSize.x,
                   (m_height + blockSize.y - 1) / blockSize.y);

	kernelRender<<<numBlocks, blockSize>>>(m_width, m_height, d_imageData_, d_spheres_, scene.spheres.size(),
        m_materials, camera.getPosition(), d_rayDirs);

	hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
		std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << "\n";
        return;
    }

    hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
        std::cerr << "CUDA kernel synchronization error: " << hipGetErrorString(err) << "\n";
		return;
	}

    err = hipMemcpy(h_imageData_, d_imageData_, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
	    std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
		return;
	}

    m_image->setData(h_imageData_);
	hipFree(d_rayDirs);
}

void Renderer::allocateDeviceMemory(const Scene& scene)
{
	if (d_spheres_)
		hipFree(d_spheres_);

	size_t numSpheres = scene.spheres.size();
	hipError_t err = hipMalloc(&d_spheres_, numSpheres * sizeof(Sphere));
	if (err != hipSuccess)
		std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";

	err = hipMemcpy(d_spheres_, scene.spheres.data(), numSpheres * sizeof(Sphere), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";

	if (m_materials)
		hipFree(m_materials);

	err = hipMalloc(&m_materials, sizeof(Material));
    if (err != hipSuccess)
		std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << "\n";

	err = hipMemcpy(m_materials, &scene.materials, sizeof(Material), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << "\n";
}

void Renderer::freeDeviceMemory()
{
	if (d_spheres_)
	{
        hipFree(d_spheres_);
        d_spheres_ = nullptr;
	}

	if (m_materials)
	{
	    hipFree(m_materials);
		m_materials = nullptr;
	}
}

__global__ void kernelRender(uint32_t width, uint32_t height, uint32_t* imageData, const Sphere* spheres,
    size_t numSpheres, const Material* material, const glm::vec3 camPos, const glm::vec3* rayDirs)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
	{
		const glm::vec4 color = Renderer::perPixel(x, y, width, height, spheres, numSpheres, material, camPos, rayDirs);
		imageData[x + y * width] = colorUtils::vec4ToRGBA(color);
    }
}

__device__ Renderer::HitRecord Renderer::traceRay(const Ray& ray, const Sphere* spheres, size_t numSpheres)
{
    int closestSphere = -1;
    float tmin = FLT_MAX;

	for (size_t i = 0; i < numSpheres; i++)
    {
        const auto& [center, radius, id] = spheres[i];

		glm::vec3 oc = ray.origin - center;

		const float a = glm::dot(ray.direction, ray.direction);
		const float b = 2.0f * glm::dot(oc, ray.direction);
		const float c = glm::dot(oc, oc) - radius * radius;
		const float discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0f)
			continue;

		float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
		float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
		const float t = t0 < t1 ? t0 : t1;

		if (t > 0.0f && t < tmin)
        {
	        tmin = t;
            closestSphere = static_cast<int>(i);
		}
	}

    if (closestSphere < 0)
        return rayMiss(ray);

	return rayHit(ray, tmin, closestSphere, spheres);
}

__device__ glm::vec4 Renderer::perPixel(uint32_t x, uint32_t y, uint32_t width, uint32_t height, const Sphere* spheres,
    size_t numSpheres, const Material* material, const glm::vec3& camPos, const glm::vec3* rayDirs)
{
    Ray ray;
    ray.origin = camPos;
    ray.direction = rayDirs[x + y * width];

    glm::vec3 light(0.0f);
    glm::vec3 throughput(1.0f);

	uint32_t seed = x + y * width;

    constexpr int bounces = 1;
    for (int i = 0; i < bounces; i++)
    {
        seed += i;

		HitRecord ht = traceRay(ray, spheres, numSpheres);
        if (ht.t < 0.0f)
        {
			auto missColor = glm::vec3(0.6f, 0.7f, 0.9f);
			light += missColor;
            break;
        }
		const auto& [center, radius, id] = spheres[ht.id];

        throughput *= material->albedo;

		ray.origin = ht.worldNormal + ht.normal * 0.0001f;
		ray.direction = glm::normalize(glm::reflect(ray.direction, ht.normal));
    }
    return { light, 1.0f };
}

__device__ Renderer::HitRecord Renderer::rayMiss(const Ray& ray)
{
    HitRecord ht;
	ht.t = -1.0f;
    return ht;
}

__device__ Renderer::HitRecord Renderer::rayHit(const Ray& ray, float tmin, int index, const Sphere* spheres)
{
    HitRecord ht;
    ht.t = tmin;
    ht.id = index;

	const glm::vec3 origin = ray.origin - spheres[index].center;
    ht.worldNormal = origin + ray.direction * tmin;
	ht.normal = glm::normalize(ht.worldNormal);

	ht.worldNormal += spheres[index].center;

    return ht;
}