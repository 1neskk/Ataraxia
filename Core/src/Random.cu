#include "hip/hip_runtime.h"
#include "Random.h"

#include <>

namespace Random
{
    __global__ void initRandomStates(hiprandState* state, unsigned long long seed)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        Random::Init(state, seed, id);
    }
}
